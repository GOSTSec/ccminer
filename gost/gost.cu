#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_streebog.h"
}

#include "miner.h"
#include "cuda_helper.h"

#include <stdio.h>
#include <memory.h>

#define NBN 2
static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

// GOST CPU Hash
extern "C" void gosthash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[128] = { 0 };

	sph_gost512_context ctx_gost1;
	sph_gost256_context ctx_gost2;	

	sph_gost512_init(&ctx_gost1);
	sph_gost512(&ctx_gost1, (const void*) input, 80);
	sph_gost512_close(&ctx_gost1, (void*) hash);

	sph_gost256_init(&ctx_gost2);
	sph_gost256(&ctx_gost2, (const void*)hash, 64);
	sph_gost256_close(&ctx_gost1, (void*) hash);

	memcpy(output, hash, 32);
}

extern void gost_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void gost_hash_32(int thr_id, uint32_t threads, uint32_t *d_hash);

//#define _DEBUG
#define _DEBUG_PREFIX "gost"
#include "cuda_debug.cuh"

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_gost(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	const int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 19 : 18; // 2^18 = 262144 cuda threads
	if (device_sm[dev_id] >= 600) intensity = 20;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0xf;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 80 * throughput), -1);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&d_hash[thr_id][k], pdata[k]);

	do {
		int order = 0;

		// Hash with CUDA
		gost_hash_64(thr_id, throughput, d_hash[thr_id]);
		TRACE("gost64   :");
		gost_hash_32(thr_id, throughput, d_hash[thr_id]);
		TRACE("gost32   :");

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&d_hash[thr_id][19], work->nonces[0]);
			gosthash(vhash, d_hash[thr_id]);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] =cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (work->nonces[1] != 0) {
					be32enc(&d_hash[thr_id][19], work->nonces[1]);
					sibhash(vhash, d_hash[thr_id]);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;

	return 0;
}

